#include "hip/hip_runtime.h"
#include "../include/lbp_cuda.cuh"
#include <iostream>

/*#define TILE_SIZE (34*34)
__global__ void _lbp_kernel_t(int* image, int* result, int rows, int cols){
    __shared__ unsigned int tile[TILE_SIZE];
    const int r = blockIdx.y*blockDim.y + threadIdx.y;
    const int c = blockIdx.x*blockDim.x + threadIdx.x;

    if(r < rows && c < cols){
        const int padded_local_pos = (threadIdx.y+1)*(blockDim.x+2) + (threadIdx.x+1);
        const int global_pos = r*cols + c;
        const int global_padded_pos = (r+1)*(cols+2) + (c+1);
        unsigned int lbp_value = 0;

        tile[padded_local_pos] = image[global_padded_pos];

        if(threadIdx.y == 0){
            tile[(threadIdx.y)*(blockDim.x+2) + (threadIdx.x+1)] = image[(r)*(cols+2) + (c+1)];
        }else if (threadIdx.y == blockDim.y)
        {
            tile[(threadIdx.y+2)*(blockDim.x+2) + (threadIdx.x+1)] = image[(r+2)*(cols+2) + (c+1)];
        }

        if(threadIdx.x == 0){
            tile[(threadIdx.y+1)*(blockDim.x+2) + (threadIdx.x)] = image[(r+1)*(cols+2) + (c)];
        }else if (threadIdx.x == blockDim.x)
        {
            tile[(threadIdx.y+1)*(blockDim.x+2) + (threadIdx.x+2)] = image[(r+1)*(cols+2) + (c+2)];
        }

        unsigned int center = tile[padded_local_pos];
        
        __syncthreads();

        lbp_value |= (tile[(threadIdx.y)*(blockDim.x+2) + (threadIdx.x)] >= center) << 7;
        lbp_value |= (tile[(threadIdx.y)*(blockDim.x+2) + (threadIdx.x+1)] >= center) << 6;
        lbp_value |= (tile[(threadIdx.y)*(blockDim.x+2) + (threadIdx.x+2)] >= center) << 5;
        lbp_value |= (tile[(threadIdx.y+1)*(blockDim.x+2) + (threadIdx.x+2)] >= center) << 4;
        lbp_value |= (tile[(threadIdx.y+2)*(blockDim.x+2) + (threadIdx.x+2)] >= center) << 3;
        lbp_value |= (tile[((threadIdx.y+2)*(blockDim.x+2) + (threadIdx.x+1))] >= center) << 2;
        lbp_value |= (tile[(threadIdx.y+2)*(blockDim.x+2) + (threadIdx.x)] >= center) << 1;
        lbp_value |= (tile[(threadIdx.y+1)*(blockDim.x+2) + (threadIdx.x)] >= center) << 0;
        

        result[global_pos] = lbp_value;
    }
}*/

__global__ void _lbp_kernel(int* image, int* result, int rows, int cols){
    const int r = blockIdx.y*blockDim.y + threadIdx.y;
    const int c = blockIdx.x*blockDim.x + threadIdx.x;

    if(r < rows && c < cols){
        const int pos = r*cols + c;
        const int padded_pos = (r+1)*(cols+2) + (c+1);
        unsigned int lbp_value = 0;

        const unsigned int center = image[padded_pos];

        lbp_value |= (image[r*(cols+2) + c] >= center) << 7;
        lbp_value |= (image[r*(cols+2) + (c+1)] >= center) << 6;
        lbp_value |= (image[r*(cols+2) + (c+2)] >= center) << 5;
        lbp_value |= (image[(r+1)*(cols+2) + (c+2)] >= center) << 4;
        lbp_value |= (image[(r+2)*(cols+2) + (c+2)] >= center) << 3;
        lbp_value |= (image[(r+2)*(cols+2) + (c+1)] >= center) << 2;
        lbp_value |= (image[(r+2)*(cols+2) + c] >= center) << 1;
        lbp_value |= (image[(r+1)*(cols+2) + c] >= center) << 0;
        

        result[pos] = lbp_value;
    }
}

__global__ void _lbp_h_kernel(int* image, unsigned int* histogram, int rows, int cols){
    __shared__ unsigned int shared_hist[256];

    const int r = blockIdx.y*blockDim.y + threadIdx.y;
    const int c = blockIdx.x*blockDim.x + threadIdx.x;
    const int pos = r*cols + c;

    if(pos < 256){
        atomicExch(&shared_hist[pos], 0);
    }

    __syncthreads();

    if(r < rows && c < cols){
        const int padded_pos = (r+1)*(cols+2) + (c+1);
        unsigned int lbp_value = 0;

        const unsigned int center = image[padded_pos];

        lbp_value |= (image[r*(cols+2) + c] >= center) << 7;
        lbp_value |= (image[r*(cols+2) + (c+1)] >= center) << 6;
        lbp_value |= (image[r*(cols+2) + (c+2)] >= center) << 5;
        lbp_value |= (image[(r+1)*(cols+2) + (c+2)] >= center) << 4;
        lbp_value |= (image[(r+2)*(cols+2) + (c+2)] >= center) << 3;
        lbp_value |= (image[(r+2)*(cols+2) + (c+1)] >= center) << 2;
        lbp_value |= (image[(r+2)*(cols+2) + c] >= center) << 1;
        lbp_value |= (image[(r+1)*(cols+2) + c] >= center) << 0;
        
        atomicAdd(&shared_hist[lbp_value], 1);
    }

    __syncthreads();

    if(pos <256){
        atomicAdd(&histogram[pos], shared_hist[pos]);
    }
}

__global__ void _histogram_kernel(int* lbp_image, unsigned int* histogram, int rows, int cols){
    __shared__ unsigned int shared_hist[256];

    const int r = blockIdx.y*blockDim.y + threadIdx.y;
    const int c = blockIdx.x*blockDim.x + threadIdx.x;
    const int pos = r*cols + c;

    if(pos < 256){
        atomicExch(&shared_hist[pos], 0);
    }

    if(r < rows && c < cols){
        atomicAdd(&shared_hist[lbp_image[pos]], 1);
    }

    __syncthreads();

    //__shared__ unsigned int counter;
    //counter = 0;
    
    //int count = atomicAdd(&counter, 1);
    if(pos <256){
        atomicAdd(&histogram[pos], shared_hist[pos]);
    }
}

unsigned int* _histogram_cuda(int* lbp_image, int rows, int cols){
    unsigned int* histogram = new unsigned int[256];

    for(int i=0; i<rows; i++){
        for(int j=0; j<cols; j++){
            histogram[lbp_image[(i*cols) + j]] += 1;
        }
    }

    return histogram; 
}

results get_LBP_hist_cuda(int* image, int rows, int cols){
    int* padded_img = new int[(rows+2)*(cols+2)];

    for(int i=0; i<rows+2; i++){
        for(int j=0; j<cols+2; j++){
            if(i == 0 || i == rows+1 || j == 0 || j == cols+1) 
                padded_img[(i*cols) + j] = 0;
            else
                padded_img[((i)*cols) + (j)] = image[((i-1)*cols) + (j-1)];
        }
    }

    unsigned int* histogram = new unsigned int[256];


    int* d_img;
    unsigned int* d_hist;
    int padded_size = (rows+2)*(cols+2)*sizeof(int); 
    int s_lbp = 1024;
    dim3 dimBlock_lbp(s_lbp, s_lbp, 1);
    dim3 dimGrid_lbp(ceil(cols/s_lbp), (rows/s_lbp), 1);

    hipMalloc((void **) &d_img, padded_size);
    hipMemcpy(d_img, padded_img, padded_size, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_hist, 256*sizeof(unsigned int));
    hipMemset(d_hist, 0, 256*sizeof(unsigned int));
    
    auto t1 = std::chrono::high_resolution_clock::now();
    _lbp_h_kernel<<<dimGrid_lbp, dimBlock_lbp>>>(d_img, d_hist, rows, cols);
    auto err = hipDeviceSynchronize();
    auto t2 = std::chrono::high_resolution_clock::now();
    
    if (err != hipSuccess) {
        printf("CUDA error during kernel launch: %s\n", hipGetErrorString(err));
        hipFree(d_img);
        hipFree(d_hist);
        exit(EXIT_FAILURE);
    }else{
        hipMemcpy(histogram, d_hist, 256*sizeof(unsigned int), hipMemcpyDeviceToHost);

        hipFree(d_img);
        hipFree(d_hist);

        results res;
        res.histogram = histogram;
        res.time = t2 - t1;

        return res;
    }
}