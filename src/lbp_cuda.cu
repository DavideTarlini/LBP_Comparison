#include "hip/hip_runtime.h"
#include "../include/lbp_cuda.cuh"
#include <iostream>

__global__ void _lbp_h_kernel(int* image, unsigned int* histogram, int rows, int cols){
    __shared__ unsigned int shared_hist[256];

    const int r = blockIdx.y*blockDim.y + threadIdx.y;
    const int c = blockIdx.x*blockDim.x + threadIdx.x;

    if((threadIdx.y*blockDim.x) + threadIdx.x < 256){
        shared_hist[(threadIdx.y*blockDim.x) + threadIdx.x] = 0;
    }
    __syncthreads();

    if(r < rows && c < cols){
        const int padded_pos = (r+1)*(cols+2) + (c+1);
        unsigned int lbp_value = 0;

        const unsigned int center = image[padded_pos];
  
        lbp_value |= (image[r*(cols+2) + c] >= center) << 7;
        lbp_value |= (image[r*(cols+2) + (c+1)] >= center) << 6;
        lbp_value |= (image[r*(cols+2) + (c+2)] >= center) << 5;
        lbp_value |= (image[(r+1)*(cols+2) + (c+2)] >= center) << 4;
        lbp_value |= (image[(r+2)*(cols+2) + (c+2)] >= center) << 3;
        lbp_value |= (image[(r+2)*(cols+2) + (c+1)] >= center) << 2;
        lbp_value |= (image[(r+2)*(cols+2) + c] >= center) << 1;
        lbp_value |= (image[(r+1)*(cols+2) + c] >= center) << 0;

        atomicAdd(&shared_hist[lbp_value], 1);
    }

    __syncthreads();
    
    if((threadIdx.y*blockDim.x) + threadIdx.x < 256){
        atomicAdd(&histogram[(threadIdx.y*blockDim.x) + threadIdx.x], shared_hist[(threadIdx.y*blockDim.x) + threadIdx.x]);
    }
}

results get_LBP_hist_cuda(int* image, int rows, int cols){
    int* padded_img = new int[(rows+2)*(cols+2)];

    for(int i=0; i<rows+2; i++){
        for(int j=0; j<cols+2; j++){
            if(i == 0 || i == rows+1 || j == 0 || j == cols+1) 
                padded_img[(i*(cols+2)) + j] = 0;
            else
                padded_img[((i)*(cols+2)) + (j)] = image[((i-1)*cols) + (j-1)];
        }
    }

    unsigned int* histogram = new unsigned int[256];
    int* d_img;
    unsigned int* d_hist;
    int padded_size = (rows+2)*(cols+2)*sizeof(int); 
    int s_lbp = 32;
    std::cout<< ceil(float(cols)/s_lbp) << "  " << ceil(float(rows)/s_lbp) << "\n\n";
    dim3 dimBlock_lbp(s_lbp, s_lbp, 1);
    dim3 dimGrid_lbp(ceil(float(cols)/s_lbp), ceil(float(rows)/s_lbp), 1);

    hipMalloc((void **) &d_img, padded_size);
    hipMemcpy(d_img, padded_img, padded_size, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_hist, 256*sizeof(unsigned int));
    hipMemset(d_hist, 0, 256*sizeof(unsigned int));
    
    auto t1 = std::chrono::high_resolution_clock::now();
    _lbp_h_kernel<<<dimGrid_lbp, dimBlock_lbp>>>(d_img, d_hist, rows, cols);
    auto err = hipDeviceSynchronize();
    auto t2 = std::chrono::high_resolution_clock::now();
    
    if (err != hipSuccess) {
        printf("CUDA error during kernel launch: %s\n", hipGetErrorString(err));
        hipFree(d_img);
        hipFree(d_hist);
        delete[] padded_img;

        exit(EXIT_FAILURE);
    }else{
        hipMemcpy(histogram, d_hist, 256*sizeof(unsigned int), hipMemcpyDeviceToHost);

        hipFree(d_img);
        hipFree(d_hist);
        delete[] padded_img;

        results res;
        res.histogram = histogram;
        res.time = t2 - t1;

        return res;
    }
}